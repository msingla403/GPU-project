#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
using namespace std;

#define pairi pair<int,int>
#define ve vector
#define vi vector<int>
#define f first
#define s second
#define t third

// class trio {
//  public:
// 	int first, second, third;

// 	void print() {
// 		cout << first << " " << second << " " << third << endl;
// 	}

// 	trio(){}

// 	trio(int a, int b, int c) {
// 		first = a;
// 		second = b;
// 		third = c;
// 	}

// 	class compare {
// 	 public:
// 		bool operator()(const trio &a, const trio &b) const {
// 			if (a.f == b.f){
// 				if(a.s==b.s)
// 					return a.t<b.t;
// 				return a.s<b.s;
// 			}
// 			return a.f > b.f;
// 		}
// 	};

// };

// float J(ve<vi>&S, int i, int j){
// 	float ans = 0.0;
// 	int count=0;

// 	for(int k=0;k<S[i].size();++k){
// 		if(S[i][k]!=0 && S[j][k]!=0){
// 			ans++;
// 		}
// 		if(S[i][k]!=0 || S[j][k]!=0){
// 			count++;
// 		}
// 	}
// 	return ans/count;
// }

// class mkDSU{
//  public:
// 	vector<int>id,size,deleted,nclusters,threshold_size;

// 	mkDSU(int n,int threshold){
// 		id.resize(n);
// 		size.resize(n);
// 		deleted.resize(n);

// 		FOR(i,0,n){
// 			id[i]=i;
// 			size[i]=1;
// 			deleted[i]=0;
// 		}
// 		nclusters=n-1;
// 		threshold_size=threshold;
// 	}

// 	int find(int a){
// 		int p=a,t;
// 		while(id[p]!=p)
// 			p=id[p];

// 		while(p!=a){
// 			t=id[a];
// 			id[a]=p;
// 			a=t;
// 		}
// 		return a;
// 	}

// 	void union_(set<pairi>& candidate_pairs, ve<vi>& S){

// 		set<trio,trio::compare()> sim_queue;

// 		for(auto it:candidate_pairs)
// 			sim_queue.insert({J(S,it.f,it.s),it.f,it.s});

// 		while(sim_queue.size() && nclusters>0){
// 			auto it = sim_queue.begin();
// 			trio temp =  *it;
// 			sim_queue.erase(it);

			
			

// 			if(i==id[i] && j==id[j]){
// 				if(deleted[i] || deleted[j])
// 					continue;

// 				if(size[i]<size[j]){
// 					id[i] = j;
// 					nclusters--;

// 					size[j] += size[i];

// 					if(size[j]>=threshold_size){
// 						deleted[j]=true;
// 						nclusters--;
// 					}

// 				}
// 				else{
// 					id[j] = i;
// 					clusters--;

// 					size[i] += size[j];

// 					if(size[i]>=threshold_size){
// 						deleted[i] = true;
// 						nclusters--;
// 					}

// 				}

// 			}


// 			else{ 

// 				int c1 = find(temp.s);
// 				int c2 = find(temp.t);

// 				if(deleted[c1] || deleted[c2] || c1==c2)
// 					continue;

// 				if(candidate_pairs.find({temp.s,temp.t})==candidate_pairs.end()){
// 					sim_queue.insert({J(S,c1,c2),min(c1,c2),max(c1,c2)});
// 					candidate_pairs.insert({min(c1,c2),max(c1,c2)});
// 				}
// 			}

// 		}
		
// 	}

// 	vi order_clusters(){
// 		map<int,vi> clusters;

// 		for(int i=0;i<n;++i){
// 			clusters[find(i)].push_back(i);
// 		}

// 		vi ans;

// 		for(auto it:clusters){
// 			for(auto ut:it){
// 				ans.push_back(ut);
// 			}
// 		}

// 		return ans;
// 	}
// };





// vi reordered_rows(ve<vi>&S){
// 	int n = S.size();

// 	set<pairi> candidate_pairs = LSH(S,5,5);


// 	mkDSU dsu(n);

// 	dsu.union_(candidate_pairs);

// 	vi ans = dsu.order_clusters();

// 	return ans;
// }


// #define PANEL_SIZE 3

// __global__ void SPMM(int * tile_row_ptr, int * panel_ptr, int * col_val, int * col_idx){

// 	int row_panel_id = blockIdx.x;
// 	int row_id = threadIdx.x/32;
// 	int thread_no = threadIdx.x%32;

// 	int num_tiles = panel_ptr[row_panel_id+1] - panel_ptr[row_panel_id];

// 	int ptr = panel_ptr[row_panel_id]*PANEL_SIZE + row_id*num_tiles;

// 	for(int i=0;i<num_tiles;++i){

// 		int low = tile_row_ptr[ptr+i];
// 		int high = tile_row_ptr[ptr+i+1];

// 		if(high>low){
// 			int j=low;
// 			O[row_id][thread_no] += col_val[j] * D[col_idx[j]][thread_no];
// 		}
// 	}
// }

// __global__ void ASPT_dense(int * panel_ptr, int * col_val, int * col_idx ){

// 	int row_panel_id = blockIdx.x;
// 	int row_id = threadIdx.x/32;
// 	int thread_no = threadIdx.x%32;

// 	int num_tiles = panel_ptr[row_panel_id+1] - panel_ptr[row_panel_id];

// 	int ptr = panel_ptr[row_panel_id]*PANEL_SIZE + row_id*num_tiles;

// 	__shared__ int map_tiles[(num_tiles-1)*PANEL_SIZE];
// 	__shared__ int shared_D[num_tiles-1][32];

// 	if(thread_no==0){
// 		for(int i=0;i<num_tiles-1;++i){

// 			int low = tile_row_ptr[ptr+i];
// 			int high = tile_row_ptr[ptr+i+1];

// 			if(high>low){
// 				map_tiles[i]=col_idx[low];
// 			}
// 		}

// 	}

// 	__syncthreads();


	

// 	__syncthreads();

// 	for(int i=0;i<num_tiles;++i){

// 		int low = tile_row_ptr[i+ptr];
// 		int high = tile_row_ptr[i+ptr+1];


// 		for(int j=low;j<=high;++j){
// 			O[row_id][thread_no] += col_val[j] * D[col_idx[j]][thread_no];
// 		}
// 	}
// }

__device__ __host__ int hashFn(int* data, int bsize)
{
	int res = bsize;
	for(int i=0; i<bsize; i++)
	{
		res ^= data[i] + 0x9e3779b9 + (res<<6) + (res>>2);
	}
	return abs(res);
}

__global__ void getSig(int *rowptr, int *colidx, int* perms, int* sigs, int siglen, int n)
{
	int idx =  blockIdx.x*blockDim.x + threadIdx.x;
	
	// if(idx == 0)
	// {
	// 	for(int i=0; i<n*siglen; i++)
	// 		printf("%d ", perms[i]);
	// 	printf("\n");
	// }

	if(idx <n)
	{
		for(int k=0; k<siglen; k++)
		{	
			int smallest = INT_MAX;
			for(int j=rowptr[idx]; j<rowptr[idx+1]; j++)
			{

				smallest = min(smallest, perms[k*n + colidx[j]]);
			}
			sigs[idx*siglen + k] = smallest;
		}
		// for(int i=0; i<siglen; i++)
		// {
		// 	printf("%d %d\n", idx, sigs[idx*siglen + i]);
		// }
	}  	
}

__global__ void getBuckets(int *sigs, int *res, int n, int siglen, int bsize, int numbuckets)
{
	int idx =  blockIdx.x*blockDim.x + threadIdx.x;

	if(idx < n)
	{
		int num_bands = siglen/bsize;
		for(int i=0; i<num_bands; i++)
		{
			int bkt = hashFn(&sigs[idx*siglen + i*bsize], bsize);
			res[idx*num_bands + i] = bkt%numbuckets;
		}
	}
}

set<pairi> LSH(vi &rowptr, vi &colidx, int siglen, int bsize, int numbuckets){
	int n = rowptr.size() - 1;

	int hperms[n*siglen];
	for(int k=0; k<siglen; k++)
	{
		vi perm(n);
		for(int i=0; i<n; i++)
		perm[i] = i;
		
		random_shuffle(perm.begin(), perm.end());
		copy(perm.begin(), perm.end(), &hperms[n*k]);		
	}

	int *drowptr;
	int *dcolidx;
	int *dperms;
	int *dsigs;
	hipMalloc(&drowptr, rowptr.size()*sizeof(int));
	hipMalloc(&dcolidx, colidx.size()*sizeof(int));
	hipMalloc(&dperms, n*siglen*sizeof(int));
	hipMalloc(&dsigs, n*siglen*sizeof(int));

	hipMemcpy(drowptr, &rowptr[0], rowptr.size()*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dcolidx, &colidx[0], colidx.size()*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dperms, hperms, n*siglen*sizeof(int), hipMemcpyHostToDevice);
	
	getSig<<< (n+1023/1024), 1024>>> (drowptr, dcolidx, dperms, dsigs, siglen, n);

	int sigs[n*siglen];
	hipMemcpy(sigs, dsigs, n*siglen*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(drowptr);
	hipFree(dcolidx);
	hipFree(dperms);
	// cudaFree(dsigs);

	// for(int i=0; i<n; i++)
	// {
	// 	for(int k=0; k<siglen; k++)
	// 		cout << sigs[i*siglen + k] << " ";
	// 	cout << endl;
	// }
	// cout << endl;
	
	int num_bands = siglen/bsize;
	int *dbucks;
	hipMalloc(&dbucks, n*num_bands*sizeof(int));

	getBuckets<<<(n+1023)/1024, 1024>>>(dsigs, dbucks, n, siglen, bsize, numbuckets);
	int hbucks[n*num_bands];
	hipMemcpy(hbucks, dbucks, n*num_bands*sizeof(int), hipMemcpyDeviceToHost);
	
	vector<set<int>> buckets(numbuckets);
	for(int i=0; i<n; i++)
	{
		for(int j=0; j<num_bands; j++)
		{
			int idx = hbucks[i*num_bands + j];
			buckets[idx].insert(i);
		}
	}

	set<pairi> result;
	for(auto s: buckets)
	{
		vi temp(s.begin(), s.end());
		for(int i=0; i<temp.size(); i++)
		{
			for(int j=i+1; j<temp.size(); j++)
			{
				result.insert(make_pair(temp[i], temp[j]));
			}
		}
	}
	return result;
}

int main()
{
	// int n = 6;
	// int m = 6;
	vi rowptr{0,2,5,7,8,11,13};
	vi colidx{0,4,1,3,5,2,4,1,0,3,4,2,5};
	
	set<pairi> candidates = LSH(rowptr, colidx, 6, 2, 6);

	for(auto i:candidates)
	{
		cout << i.f << " " << i.s << endl;
	}
}



