#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
using namespace std;

#define pairi pair<int,int>
#define ve vector
#define vi vector<int>
#define f first
#define s second
#define t third



__device__ __host__ int hashFn(int* data, int bsize)
{
	int res = bsize;
	for(int i=0; i<bsize; i++)
	{
		res ^= data[i] + 0x9e3779b9 + (res<<6) + (res>>2);
	}
	return abs(res);
}

__global__ void getSig(int *rowptr, int *colidx, int* perms, int* sigs, int siglen, int n)
{
	int idx =  blockIdx.x*blockDim.x + threadIdx.x;
	
	// if(idx == 0)
	// {
	// 	for(int i=0; i<n*siglen; i++)
	// 		printf("%d ", perms[i]);
	// 	printf("\n");
	// }

	if(idx <n)
	{
		for(int k=0; k<siglen; k++)
		{	
			int smallest = INT_MAX;
			for(int j=rowptr[idx]; j<rowptr[idx+1]; j++)
			{

				smallest = min(smallest, perms[k*n + colidx[j]]);
			}
			sigs[idx*siglen + k] = smallest;
		}
		// for(int i=0; i<siglen; i++)
		// {
		// 	printf("%d %d\n", idx, sigs[idx*siglen + i]);
		// }
	}  	
}

__global__ void getBuckets(int *sigs, int *res, int n, int siglen, int bsize, int numbuckets)
{
	int idx =  blockIdx.x*blockDim.x + threadIdx.x;

	if(idx < n)
	{
		int num_bands = siglen/bsize;
		for(int i=0; i<num_bands; i++)
		{
			int bkt = hashFn(&sigs[idx*siglen + i*bsize], bsize);
			res[idx*num_bands + i] = bkt%numbuckets;
		}
	}
}

set<pairi> LSH(vi &rowptr, vi &colidx, int siglen, int bsize, int numbuckets){
	int n = rowptr.size() - 1;

	int hperms[n*siglen];
	for(int k=0; k<siglen; k++)
	{
		vi perm(n);
		for(int i=0; i<n; i++)
		perm[i] = i;
		
		random_shuffle(perm.begin(), perm.end());
		copy(perm.begin(), perm.end(), &hperms[n*k]);		
	}

	int *drowptr;
	int *dcolidx;
	int *dperms;
	int *dsigs;
	hipMalloc(&drowptr, rowptr.size()*sizeof(int));
	hipMalloc(&dcolidx, colidx.size()*sizeof(int));
	hipMalloc(&dperms, n*siglen*sizeof(int));
	hipMalloc(&dsigs, n*siglen*sizeof(int));

	hipMemcpy(drowptr, &rowptr[0], rowptr.size()*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dcolidx, &colidx[0], colidx.size()*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dperms, hperms, n*siglen*sizeof(int), hipMemcpyHostToDevice);
	
	getSig<<< (n+1023/1024), 1024>>> (drowptr, dcolidx, dperms, dsigs, siglen, n);

	int sigs[n*siglen];
	hipMemcpy(sigs, dsigs, n*siglen*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(drowptr);
	hipFree(dcolidx);
	hipFree(dperms);
	// hipFree(dsigs);

	// for(int i=0; i<n; i++)
	// {
	// 	for(int k=0; k<siglen; k++)
	// 		cout << sigs[i*siglen + k] << " ";
	// 	cout << endl;
	// }
	// cout << endl;
	
	int num_bands = siglen/bsize;
	int *dbucks;
	hipMalloc(&dbucks, n*num_bands*sizeof(int));

	getBuckets<<<(n+1023)/1024, 1024>>>(dsigs, dbucks, n, siglen, bsize, numbuckets);
	int hbucks[n*num_bands];
	hipMemcpy(hbucks, dbucks, n*num_bands*sizeof(int), hipMemcpyDeviceToHost);
	
	vector<set<int>> buckets(numbuckets);
	for(int i=0; i<n; i++)
	{
		for(int j=0; j<num_bands; j++)
		{
			int idx = hbucks[i*num_bands + j];
			buckets[idx].insert(i);
		}
	}

	set<pairi> result;
	for(auto s: buckets)
	{
		vi temp(s.begin(), s.end());
		for(int i=0; i<temp.size(); i++)
		{
			for(int j=i+1; j<temp.size(); j++)
			{
				result.insert(make_pair(temp[i], temp[j]));
			}
		}
	}
	return result;
}

#define PANEL_SIZE 3
#define DENSE_THRESHOLD 2

__global__ void SPMM(int * tile_row_ptr, int * panel_ptr, int * col_val, int * col_idx){

	int row_panel_id = blockIdx.x;
	int row_id = threadIdx.x/32;
	int thread_no = threadIdx.x%32;

	int num_tiles = panel_ptr[row_panel_id+1] - panel_ptr[row_panel_id];

	int ptr = panel_ptr[row_panel_id]*PANEL_SIZE + row_id*num_tiles;

	for(int i=0;i<num_tiles;++i){

		int low = tile_row_ptr[ptr+i];
		int high = tile_row_ptr[ptr+i+1];

		if(high>low){
			int j=low;
			O[row_id][thread_no] += col_val[j] * D[col_idx[j]][thread_no];
		}
	}
}

__global__ find_dense(int *col_ptr, int* row_idx, int *isdense, int nr, int nc)
{	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < (nr/PANEL_SIZE)*nc)
	{
		int panel_id = idx/nc;
		int col_id = idx%nc;
		
		int counter = 0;
		for(int i=col_ptr[col_id]; i<col_ptr[col_id+1]; i++)
		{
			if(row_index[i] >= panel_id*PANEL_SIZE && row_idx[i] < (panel_id+1)*PANEL_SIZE)
				counter++;
		}

		if(counter >= DENSE_THRESHOLD)
			isdense[idx] = 1;
		else
			isdense[idx] = 0;
	}

}

int main(int argc, char** argv)
{
	char* inputfilename = argv[1];
	FILE *fp;
	fp = fopen(inputfilename, "r");
	
	int nr, nc, ne;
	fscanf(fp, "%d %d %d", &nr, &nc, &ne);

	int row_ptr[nr+1];
	int col_idx[ne];
	for(int i=0; i<=nr; i++)
		row_ptr[i] = 0;

	int col_val[ne];


	int col_ptr[nc+1];
	int row_idx[ne];
	for(int i=0; i<=nr; i++)
		col_ptr[i] = 0;
		
	int r, c;
	
	for(int i=0; i<ne; i++)
	{
		fscanf(fp, "%d %d", &r, &c);
		
		row_ptr[r]++;
		col_idx[i] = c-1;
		col_val[i] = 1;
		
		col_ptr[c]++;
		row_idx[i] = r-1;
	}
	for(int i=0; i<nr; i++)
		row_ptr[i+1] += row_ptr[i];

	for(int i=0; i<nc; i++)
		col_ptr[i+1] += col_ptr[i];
	
	// for(int i=0; i<=nr; i++)
	// 	cout << row_ptr[i] << " ";
	
	// cout <<endl;

	// for(int i=0; i<ne; i++)
	// 	cout << col_idx[i] << " ";
	// cout << endl;
	int num_panels = nr/PANEL_SIZE;
	int thr = num_panels*nc;

	int *dcol_ptr;
	int *drow_idx;
	int *is_dense;
	hipMalloc(&dcol_ptr, (nr+1)*sizeof(int));
	hipMalloc(&drow_idx, ne*sizeof(int));
	hipMalloc(&is_dense, thr*sizeof(int));

	hipMemcpy(dcol_ptr, col_ptr, (nr+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(drow_idx, row_idx, ne*sizeof(int), hipMemcpyHostToDevice);

	find_dense<<< (thr+1023)/1024, 1024>>>(dcol_ptr, drow_idx, is_dense, nr, nc);

	int isdense[thr];
	hipMemcpy(isdense, is_dense, thr*sizeof(int), hipMemcpyDeviceToHost);
	
	// int n = 6;
	// int m = 6;
	// vi rowptr{0,2,5,7,8,11,13};
	// vi colidx{0,4,1,3,5,2,4,1,0,3,4,2,5};
	
	// set<pairi> candidates = LSH(rowptr, colidx, 6, 2, 6);

	// for(auto i:candidates)
	// {
	// 	cout << i.f << " " << i.s << endl;
	// }
}



